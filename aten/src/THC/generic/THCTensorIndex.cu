
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorIndex.cu"
#else

// Check tensor dimensions for index operations, and return the slice size.
// src can be nullptr in case of indexFill: in that case it is ignored.
static ptrdiff_t THCTensor_(getSliceSize)(THCState *state, THCTensor *dst,
                                          int dim,
                                          THCudaLongTensor *index,
                                          THCTensor *src)
{
  int dstDims = THCTensor_(nDimension)(state, dst);
  int srcDims = (src == nullptr) ? dstDims : THCTensor_(nDimension)(state, src);

  THArgCheck(THCudaLongTensor_nDimension(state, index) == 1, 4,
             "expecting vector of indices");
  THArgCheck(dim >= 0 && dim < dstDims, 2, "Indexing dim is out of bounds");

  ptrdiff_t dstSliceSize = 1;
  for (int d = 0; d < dstDims; d++) {
    if (d != dim) {
      dstSliceSize *= dst->size[d];
    }
  }

  if (src == nullptr) return dstSliceSize;

  THArgCheck(dim < srcDims, 3, "Indexing dim is out of bounds");
  THArgCheck(THCudaLongTensor_nElement(state, index) == src->size[dim], 4,
             "length of src.size[dim] is not equal to length of indices");

  ptrdiff_t srcSliceSize = 1;
  bool mismatch = false;

  if (dstDims != srcDims) mismatch = true;

  for (int d = 0; d < srcDims; d++) {
    if (d != dim) {
      srcSliceSize *= src->size[d];
      if (!mismatch && dst->size[d] != src->size[d]) mismatch = true;
    }
  }

  THArgCheck(dstSliceSize == srcSliceSize, 2,
             "Source/destination tensor have different slice sizes (%ld vs %ld)",
             dstSliceSize, srcSliceSize);

  if (mismatch) {
    static bool warningShown = false;
    if (!warningShown) {
      warningShown = true;
      fprintf(stderr,
              "Warning: source/destination slices have same size but different "
              "shape for an index operation.  This behavior is deprecated.\n");
    }
  }

  return dstSliceSize;
}

// Compare the stride between adjacent slices (sliceStride) with strides in the
// other dimensions (i.e., strides *inside* each slice).
//
// - Returns true if some dimension inside the slice has lower stride than
//   sliceStride.  The simplest example is a 2-D contiguous tensor with sliceDim
//   == 0 (that is, each slice is a row).
//
//   In this case, we choose the CUDA kernel that processes the data in
//   "index-major order".  For example, if thread count equals slice size, then
//   all threads process slice #0 in lockstep, and then slice #1, and so on.
//
// - Otherwise (i.e., sliceStride has the lowest value), this function returns
//   false.  The simplest example is a 2-D contiguous tensor with sliceDim == 1
//   (each slice is a column).
//
//   In this case, we choose the CUDA kernel that processes the data in
//   "elementInSlice-major order".  For example, each thread can process element
//   #0 of every slice, and then element #1 of every slice, and so on.
bool THCTensor_(indexShouldBeMajor)(TensorInfo<real, unsigned int> &info,
                                    int sliceDim)
{
  // The stride between adjacent slices (e.g., between element #0 of slice #100
  // and element #0 of slice #101).
  unsigned int sliceStride = info.strides[sliceDim];

  for (int i = 0; i < info.dims; ++i) {
    if (i != sliceDim && info.sizes[i] > 1 && info.strides[i] < sliceStride) {
      return true;
    }
  }

  return false;
}

void THCTensor_(indexCopy)(THCState *state, THCTensor *dst, int dim, THCudaLongTensor *indices, THCTensor *src)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, dst, src));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, indices));

  int dims = THCTensor_(nDimension)(state, dst);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  dims = THCTensor_(nDimension)(state, src);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 5, CUTORCH_DIM_WARNING);
  dims = THCudaLongTensor_nDimension(state, indices);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 4, CUTORCH_DIM_WARNING);

  // The `src` is partitioned into two parts:
  // -the size of each slice we are indexing, which is the
  // total size of the tensor ignoring dimension `dim`;
  // -the number of indices we are choosing, which is the total size
  // of the tensor `indices`.
  ptrdiff_t sliceSize = THCTensor_(getSliceSize)(state, dst, dim, indices, src);
  ptrdiff_t srcTotalSize = THCTensor_(nElement)(state, src);
  int64_t dstCopyDimSize = THCTensor_(size)(state, dst, dim);

  ptrdiff_t numIndices = THCudaLongTensor_nElement(state, indices);
  cudaStream_t stream = THCState_getCurrentStream(state);
  int indContig = THCudaLongTensor_isContiguous(state, indices);

  int mpc = THCState_getCurrentDeviceProperties(state)->multiProcessorCount;

#define SMALL_INDEX(TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM) \
  indexCopySmallIndex<TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM>       \
    <<<smallIndexGrid, smallIndexBlock, 0, stream>>>(           \
      dstInfo, srcInfo, indicesInfo,                            \
      dstCopyDim, srcCopyDim, sliceSize, dstCopyDimSize);

#define LARGE_INDEX(TENSOR_TYPE, TYPE,                         \
                    DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR)   \
  indexCopyLargeIndex<TENSOR_TYPE, TYPE,                       \
                      DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR> \
    <<<largeIndexGrid, largeIndexBlock, 0, stream>>>(          \
      dstInfo, srcInfo, indicesInfo,                           \
      dstCopyDim, srcCopyDim, srcTotalSize,                    \
      (IDX_IS_MAJOR) ? sliceSize : numIndices,                 \
      dstCopyDimSize);

  dim3 smallIndexGrid(std::min(THCCeilDiv(sliceSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 smallIndexBlock(std::min(sliceSize, (ptrdiff_t)128));

  dim3 largeIndexGrid(std::min(THCCeilDiv(srcTotalSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 largeIndexBlock(std::min(srcTotalSize, (ptrdiff_t)128));

  if (THCTensor_canUse32BitIndexMath(state, dst) &&
      THCTensor_canUse32BitIndexMath(state, src) &&
      THCTensor_canUse32BitIndexMath(state, indices)) {
    TensorInfo<real, unsigned int> dstInfo =
      getTensorInfo<real, THCTensor, unsigned int>(state, dst);
    int dstCopyDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstCopyDim);

    TensorInfo<real, unsigned int> srcInfo =
      getTensorInfo<real, THCTensor, unsigned int>(state, src);
    int srcCopyDim = srcInfo.collapseDims(dim);
    srcInfo.reduceDim(srcCopyDim);

    TensorInfo<int64_t, unsigned int> indicesInfo =
      getTensorInfo<int64_t, THCudaLongTensor, unsigned int>(state, indices);
    indicesInfo.collapseDims();

    // A reasonable choice for when to have each thread iterate over
    // indices to choose
    if (numIndices <= 16) {
      if (dstInfo.dims == 1 && srcInfo.dims == 1 && indContig) {
        SMALL_INDEX(real, unsigned int, 1, 1, -2);
      } else if (dstInfo.dims == 2 && srcInfo.dims == 2 && indContig) {
        SMALL_INDEX(real, unsigned int, 2, 2, -2);
      } else if (dstInfo.dims == 3 && srcInfo.dims == 3 && indContig) {
        SMALL_INDEX(real, unsigned int, 3, 3, -2);
      } else {
        SMALL_INDEX(real, unsigned int, -1, -1, -1);
      }
    } else {
      bool indexIsMajor = THCTensor_(indexShouldBeMajor)(dstInfo, dstCopyDim);

      if (dstInfo.dims == 1 && srcInfo.dims == 1 && indContig) {
        LARGE_INDEX(real, unsigned int, 1, 1, -2, true);
      } else if (dstInfo.dims == 2 && srcInfo.dims == 2 && indContig) {
        if (indexIsMajor) {
          LARGE_INDEX(real, unsigned int, 2, 2, -2, true);
        } else {
          LARGE_INDEX(real, unsigned int, 2, 2, -2, false);
        }
      } else if (dstInfo.dims == 3 && srcInfo.dims == 3 && indContig) {
        if (indexIsMajor) {
          LARGE_INDEX(real, unsigned int, 3, 3, -2, true);
        } else {
          LARGE_INDEX(real, unsigned int, 3, 3, -2, false);
        }
      } else {
        LARGE_INDEX(real, unsigned int, -1, -1, -1, true);
      }
    }
  } else {
    TensorInfo<real, uint64_t> dstInfo =
      getTensorInfo<real, THCTensor, uint64_t>(state, dst);
    int dstCopyDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstCopyDim);

    TensorInfo<real, uint64_t> srcInfo =
      getTensorInfo<real, THCTensor, uint64_t>(state, src);
    int srcCopyDim = srcInfo.collapseDims(dim);
    srcInfo.reduceDim(srcCopyDim);

    TensorInfo<int64_t, uint64_t> indicesInfo =
      getTensorInfo<int64_t, THCudaLongTensor, uint64_t>(state, indices);
    indicesInfo.collapseDims();

    LARGE_INDEX(real, uint64_t, -1, -1, -1, true);
  }

#undef SMALL_INDEX
#undef LARGE_INDEX
}

void THCTensor_(take)(THCState *state, THCTensor *dst, THCTensor *src, THCudaLongTensor *index)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, dst, src));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, index));

  THArgCheck(THCTensor_(nDimension)(state, src) <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  THArgCheck(THCTensor_(nDimension)(state, dst) <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  THArgCheck(THCudaLongTensor_nDimension(state, index) <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  THArgCheck(!(THCTensor_(nDimension)(state, src) == 0 && THCudaLongTensor_nDimension(state, index) != 0), 2,
             "tried to take from an empty tensor");

  THCTensor_(resizeNd)(state, dst, index->_dim(), index->size, NULL);

  // dispatchTakePut only handles non-empty tensors;
  if (index->_dim() > 0) {
    dispatchTakePut<real, TensorTakeOp>(state, src, dst, index);
  }
}

static void THCTensor_(sort_indices)(THCState *state, THCudaLongTensor *index, THCTensor *src) {
  THCThrustAllocator thrustAlloc(state);

  auto index_iter = thrust::device_ptr<int64_t>(THCudaLongTensor_data(state, index));
  auto src_iter = thrust::device_ptr<real>(THCTensor_(data)(state, src));
  auto numel = THCTensor_(numel)(state, src);

  thrust::sort_by_key(
    thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
    index_iter, index_iter + numel,
    src_iter, ThrustLTOp<int64_t>());
}

void THCTensor_(put)(THCState *state, THCTensor *dst, THCudaLongTensor *index, THCTensor *src, int accumulate)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, dst, src));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, index));

  ptrdiff_t dstSize = THCTensor_(nElement)(state, dst);
  ptrdiff_t numIndices = THCudaLongTensor_nElement(state, index);
  THArgCheck(THCTensor_(nElement)(state, src) == numIndices,
    3, "src should have the same number of elements as index");

  THArgCheck(THCTensor_(nDimension)(state, dst) <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  THArgCheck(THCTensor_(nDimension)(state, src) <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  THArgCheck(THCudaLongTensor_nDimension(state, index) <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);

  if (numIndices == 0) {
    return;
  }

  if (accumulate) {
    // wrap indices so to replace negative indices
    THCudaLongTensor* sorted_index = THCudaLongTensor_new(state);
    THCudaLongTensor_resizeAs(state, sorted_index, index);
    THC_pointwiseApply2<int64_t, int64_t>(state, sorted_index, index, WrapIndexOp(dstSize));

    THCTensor* sorted_src = THCTensor_(newClone)(state, src);

    THCTensor_(sort_indices)(state, sorted_index, sorted_src);
    dispatchTakePut<real, TensorPutAccumulateOp>(state, dst, sorted_src, sorted_index);

    THCTensor_(free)(state, sorted_src);
    THCudaLongTensor_free(state, sorted_index);
  } else {
    dispatchTakePut<real, TensorPutOp>(state, dst, src, index);
  }
}

void THCTensor_(indexAdd)(THCState *state, THCTensor *dst, int dim, THCudaLongTensor *indices, THCTensor *src)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, dst, src));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, indices));

  int dims = THCTensor_(nDimension)(state, dst);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  dims = THCTensor_(nDimension)(state, src);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 5, CUTORCH_DIM_WARNING);
  dims = THCudaLongTensor_nDimension(state, indices);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 4, CUTORCH_DIM_WARNING);

  // The `src` is partitioned into two parts:
  // -the size of each slice we are indexing, which is the
  // total size of the tensor ignoring dimension `dim`;
  // -the number of indices we are choosing, which is the total size
  // of the tensor `indices`.
  ptrdiff_t sliceSize = THCTensor_(getSliceSize)(state, dst, dim, indices, src);
  ptrdiff_t srcTotalSize = THCTensor_(nElement)(state, src);
  int64_t dstAddDimSize = THCTensor_(size)(state, dst, dim);

  ptrdiff_t numIndices = THCudaLongTensor_nElement(state, indices);
  cudaStream_t stream = THCState_getCurrentStream(state);
  int indContig = THCudaLongTensor_isContiguous(state, indices);

  int mpc = THCState_getCurrentDeviceProperties(state)->multiProcessorCount;

#define SMALL_INDEX(TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM) \
  indexAddSmallIndex<TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM> \
    <<<smallIndexGrid, smallIndexBlock, 0, stream>>>(   \
      dstInfo, srcInfo, indicesInfo,                    \
      dstAddDim, srcAddDim, sliceSize, dstAddDimSize);

#define LARGE_INDEX(TENSOR_TYPE, TYPE,                        \
                    DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR)  \
  indexAddLargeIndex<TENSOR_TYPE, TYPE,                       \
                     DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR> \
    <<<largeIndexGrid, largeIndexBlock, 0, stream>>>(         \
      dstInfo, srcInfo, indicesInfo,                          \
      dstAddDim, srcAddDim, srcTotalSize,                     \
      (IDX_IS_MAJOR) ? sliceSize : numIndices,                \
      dstAddDimSize);

  dim3 smallIndexGrid(std::min(THCCeilDiv(sliceSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 smallIndexBlock(std::min(sliceSize, (ptrdiff_t)128));

  dim3 largeIndexGrid(std::min(THCCeilDiv(srcTotalSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 largeIndexBlock(std::min(srcTotalSize, (ptrdiff_t)128));

  if (THCTensor_canUse32BitIndexMath(state, dst) &&
      THCTensor_canUse32BitIndexMath(state, src) &&
      THCTensor_canUse32BitIndexMath(state, indices)) {
    TensorInfo<real, unsigned int> dstInfo =
      getTensorInfo<real, THCTensor, unsigned int>(state, dst);
    int dstAddDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstAddDim);

    TensorInfo<real, unsigned int> srcInfo =
      getTensorInfo<real, THCTensor, unsigned int>(state, src);
    int srcAddDim = srcInfo.collapseDims(dim);
    srcInfo.reduceDim(srcAddDim);

    TensorInfo<int64_t, unsigned int> indicesInfo =
      getTensorInfo<int64_t, THCudaLongTensor, unsigned int>(state, indices);
    indicesInfo.collapseDims();

    // A reasonable choice for when to have each thread iterate over
    // indices to choose
    if (numIndices <= 16) {
      if (dstInfo.dims == 1 && srcInfo.dims == 1 && indContig) {
        SMALL_INDEX(real, unsigned int, 1, 1, -2);
      } else if (dstInfo.dims == 2 && srcInfo.dims == 2 && indContig) {
        SMALL_INDEX(real, unsigned int, 2, 2, -2);
      } else if (dstInfo.dims == 3 && srcInfo.dims == 3 && indContig) {
        SMALL_INDEX(real, unsigned int, 3, 3, -2);
      } else {
        SMALL_INDEX(real, unsigned int, -1, -1, -1);
      }
    } else {
      bool indexIsMajor = THCTensor_(indexShouldBeMajor)(dstInfo, dstAddDim);

      if (dstInfo.dims == 1 && srcInfo.dims == 1 && indContig) {
        LARGE_INDEX(real, unsigned int, 1, 1, -2, true);
      } else if (dstInfo.dims == 2 && srcInfo.dims == 2 && indContig) {
        if (indexIsMajor) {
          LARGE_INDEX(real, unsigned int, 2, 2, -2, true);
        } else {
          LARGE_INDEX(real, unsigned int, 2, 2, -2, false);
        }
      } else if (dstInfo.dims == 3 && srcInfo.dims == 3 && indContig) {
        if (indexIsMajor) {
          LARGE_INDEX(real, unsigned int, 3, 3, -2, true);
        } else {
          LARGE_INDEX(real, unsigned int, 3, 3, -2, false);
        }
      } else {
        LARGE_INDEX(real, unsigned int, -1, -1, -1, true);
      }
    }
  } else {
    TensorInfo<real, uint64_t> dstInfo =
      getTensorInfo<real, THCTensor, uint64_t>(state, dst);
    int dstAddDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstAddDim);

    TensorInfo<real, uint64_t> srcInfo =
      getTensorInfo<real, THCTensor, uint64_t>(state, src);
    int srcAddDim = srcInfo.collapseDims(dim);
    srcInfo.reduceDim(srcAddDim);

    TensorInfo<int64_t, uint64_t> indicesInfo =
      getTensorInfo<int64_t, THCudaLongTensor, uint64_t>(state, indices);
    indicesInfo.collapseDims();

    LARGE_INDEX(real, uint64_t, -1, -1, -1, true);
  }

#undef SMALL_INDEX
#undef LARGE_INDEX
}

void THCTensor_(indexFill)(THCState *state, THCTensor *dst, int dim, THCudaLongTensor *indices, real val)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, dst));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, indices));
  int dims = THCTensor_(nDimension)(state, dst);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  dims = THCudaLongTensor_nDimension(state, indices);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 4, CUTORCH_DIM_WARNING);

  // The `src` is partitioned into two parts:
  // -the size of each slice we are indexing, which is the
  // total size of the tensor ignoring dimension `dim`;
  // -the number of indices we are choosing, which is the total size
  // of the tensor `indices`.
  ptrdiff_t sliceSize =
    THCTensor_(getSliceSize)(state, dst, dim, indices, nullptr);
  ptrdiff_t dstTotalSize = THCTensor_(nElement)(state, dst);
  int64_t dstFillDimSize = THCTensor_(size)(state, dst, dim);

  ptrdiff_t numIndices = THCudaLongTensor_nElement(state, indices);
  cudaStream_t stream = THCState_getCurrentStream(state);
  int indContig = THCudaLongTensor_isContiguous(state, indices);

  int mpc = THCState_getCurrentDeviceProperties(state)->multiProcessorCount;

#define SMALL_INDEX(TENSOR_TYPE, TYPE, DST_DIM, IDX_DIM)  \
  indexFillSmallIndex<TENSOR_TYPE, TYPE, DST_DIM, IDX_DIM> \
    <<<smallIndexGrid, smallIndexBlock, 0, stream>>>(   \
      dstInfo, indicesInfo,                             \
      dstFillDim, sliceSize, dstFillDimSize, val);

#define LARGE_INDEX(TENSOR_TYPE, TYPE, DST_DIM, IDX_DIM, IDX_IS_MAJOR)   \
  indexFillLargeIndex<TENSOR_TYPE, TYPE, DST_DIM, IDX_DIM, IDX_IS_MAJOR> \
    <<<largeIndexGrid, largeIndexBlock, 0, stream>>>(                    \
      dstInfo, indicesInfo,                                              \
      dstFillDim, sliceSize * numIndices,                                \
      (IDX_IS_MAJOR) ? sliceSize : numIndices,                           \
      dstFillDimSize, val);

  dim3 smallIndexGrid(std::min(THCCeilDiv(sliceSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 smallIndexBlock(std::min(sliceSize, (ptrdiff_t)128));

  dim3 largeIndexGrid(std::min(THCCeilDiv(dstTotalSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 largeIndexBlock(std::min(dstTotalSize, (ptrdiff_t)128));

  if (THCTensor_canUse32BitIndexMath(state, dst) &&
      THCTensor_canUse32BitIndexMath(state, indices)) {
    TensorInfo<real, unsigned int> dstInfo =
      getTensorInfo<real, THCTensor, unsigned int>(state, dst);
    int dstFillDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstFillDim);

    TensorInfo<int64_t, unsigned int> indicesInfo =
      getTensorInfo<int64_t, THCudaLongTensor, unsigned int>(state, indices);
    indicesInfo.collapseDims();

    // A reasonable choice for when to have each thread iterate over
    // indices to choose
    if (numIndices <= 16) {
      if (dstInfo.dims == 1 && indContig) {
        SMALL_INDEX(real, unsigned int, 1, -2);
      } else if (dstInfo.dims == 2 && indContig) {
        SMALL_INDEX(real, unsigned int, 2, -2);
      } else if (dstInfo.dims == 3 && indContig) {
        SMALL_INDEX(real, unsigned int, 3, -2);
      } else {
        SMALL_INDEX(real, unsigned int, -1, -1);
      }
    } else {
      bool indexIsMajor = THCTensor_(indexShouldBeMajor)(dstInfo, dstFillDim);

      if (dstInfo.dims == 1 && indContig) {
        LARGE_INDEX(real, unsigned int, 1, -2, true);
      } else if (dstInfo.dims == 2 && indContig) {
        if (indexIsMajor) {
          LARGE_INDEX(real, unsigned int, 2, -2, true);
        } else {
          LARGE_INDEX(real, unsigned int, 2, -2, false);
        }
      } else if (dstInfo.dims == 3 && indContig) {
        if (indexIsMajor) {
          LARGE_INDEX(real, unsigned int, 3, -2, true);
        } else {
          LARGE_INDEX(real, unsigned int, 3, -2, false);
        }
      } else {
        LARGE_INDEX(real, unsigned int, -1, -1, true);
      }
    }
  } else {
    TensorInfo<real, uint64_t> dstInfo =
      getTensorInfo<real, THCTensor, uint64_t>(state, dst);
    int dstFillDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstFillDim);

    TensorInfo<int64_t, uint64_t> indicesInfo =
      getTensorInfo<int64_t, THCudaLongTensor, uint64_t>(state, indices);
    indicesInfo.collapseDims();

    LARGE_INDEX(real, uint64_t, -1, -1, true);
  }

#undef SMALL_INDEX
#undef LARGE_INDEX
}

void THCTensor_(indexSelect)(THCState *state, THCTensor *dst, THCTensor *src, int dim, THCudaLongTensor *indices)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, dst, src, indices));

  int dims = THCTensor_(nDimension)(state, dst);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  dims = THCTensor_(nDimension)(state, src);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 3, CUTORCH_DIM_WARNING);
  dims = THCudaLongTensor_nDimension(state, indices);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 5, CUTORCH_DIM_WARNING);

  ptrdiff_t numIndices = THCudaLongTensor_nElement(state, indices);

  int srcDims = THCTensor_(nDimension)(state, src);
  cudaStream_t stream = THCState_getCurrentStream(state);

  THArgCheck(THCudaLongTensor_nDimension(state, indices) <= 1, 3,
             "Index is supposed to be an empty tensor or a vector");
  THArgCheck(dim < srcDims, 4, "Indexing dim is out of bounds");
  THArgCheck(srcDims > 0, 2, "Source tensor is empty");

  THLongStorage *newSize;

  if (numIndices == 0) {
    newSize = THCTensor_(newSizeOf)(state, src);
    THLongStorage_set(newSize, 0, numIndices);
    THCTensor_(resize)(state, dst, newSize, NULL);
    THLongStorage_free(newSize);
    return;
  }

  newSize = THCTensor_(newSizeOf)(state, src);
  THLongStorage_set(newSize, dim, numIndices);
  THCTensor_(resize)(state, dst, newSize, NULL);
  THLongStorage_free(newSize);

  int indContig = THCudaLongTensor_isContiguous(state, indices);

  // The `src` is partitioned into two parts:
  // -the size of each slice we are indexing, which is the
  // total size of the tensor ignoring dimension `dim`;
  // -the number of indices we are choosing, which is the total size
  // of the tensor `indices`.
  ptrdiff_t dstTotalSize = THCTensor_(nElement)(state, dst);
  int64_t srcSelectDimSize = THCTensor_(size)(state, src, dim);
  ptrdiff_t sliceSize = dstTotalSize / numIndices;

  int mpc = THCState_getCurrentDeviceProperties(state)->multiProcessorCount;

#define SMALL_INDEX(TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM) \
  indexSelectSmallIndex<TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM>     \
    <<<smallIndexGrid, smallIndexBlock, 0, stream>>>(           \
      dstInfo, srcInfo, indicesInfo,                            \
      dstSelectDim, srcSelectDim, sliceSize, srcSelectDimSize);

#define LARGE_INDEX(TENSOR_TYPE, TYPE,                           \
                    DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR)     \
  indexSelectLargeIndex<TENSOR_TYPE, TYPE,                       \
                        DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR> \
    <<<largeIndexGrid, largeIndexBlock, 0, stream>>>(            \
      dstInfo, srcInfo, indicesInfo,                             \
      dstSelectDim, srcSelectDim, dstTotalSize,                  \
      (IDX_IS_MAJOR) ? sliceSize : numIndices,                   \
      srcSelectDimSize);

  dim3 smallIndexGrid(std::min(THCCeilDiv(sliceSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 smallIndexBlock(std::min(sliceSize, (ptrdiff_t)128));

  dim3 largeIndexGrid(std::min(THCCeilDiv(dstTotalSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 largeIndexBlock(std::min(dstTotalSize, (ptrdiff_t)128));

  if (THCTensor_canUse32BitIndexMath(state, dst) &&
      THCTensor_canUse32BitIndexMath(state, src) &&
      THCTensor_canUse32BitIndexMath(state, indices)) {
    TensorInfo<real, unsigned int> dstInfo =
      getTensorInfo<real, THCTensor, unsigned int>(state, dst);
    int dstSelectDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstSelectDim);

    TensorInfo<real, unsigned int> srcInfo =
      getTensorInfo<real, THCTensor, unsigned int>(state, src);
    int srcSelectDim = srcInfo.collapseDims(dim);
    srcInfo.reduceDim(srcSelectDim);

    TensorInfo<int64_t, unsigned int> indicesInfo =
      getTensorInfo<int64_t, THCudaLongTensor, unsigned int>(state, indices);
    indicesInfo.collapseDims();

    // A reasonable choice for when to have each thread iterate over
    // indices to choose
    if (numIndices <= 16) {
      if (dstInfo.dims == 1 && srcInfo.dims == 1 && indContig) {
        SMALL_INDEX(real, unsigned int, 1, 1, -2);
      } else if (dstInfo.dims == 2 && srcInfo.dims == 2 && indContig) {
        SMALL_INDEX(real, unsigned int, 2, 2, -2);
      } else if (dstInfo.dims == 3 && srcInfo.dims == 3 && indContig) {
        SMALL_INDEX(real, unsigned int, 3, 3, -2);
      } else {
        SMALL_INDEX(real, unsigned int, -1, -1, -1);
      }
    } else {
      bool indexIsMajor = THCTensor_(indexShouldBeMajor)(dstInfo, dstSelectDim);

      if (dstInfo.dims == 1 && srcInfo.dims == 1 && indContig) {
        LARGE_INDEX(real, unsigned int, 1, 1, -2, true);
      } else if (dstInfo.dims == 2 && srcInfo.dims == 2 && indContig) {
        if (indexIsMajor) {
          LARGE_INDEX(real, unsigned int, 2, 2, -2, true);
        } else {
          LARGE_INDEX(real, unsigned int, 2, 2, -2, false);
        }
      } else if (dstInfo.dims == 3 && srcInfo.dims == 3 && indContig) {
        if (indexIsMajor) {
          LARGE_INDEX(real, unsigned int, 3, 3, -2, true);
        } else {
          LARGE_INDEX(real, unsigned int, 3, 3, -2, false);
        }
      } else {
        LARGE_INDEX(real, unsigned int, -1, -1, -1, true);
      }
    }
  } else {
    TensorInfo<real, uint64_t> dstInfo =
      getTensorInfo<real, THCTensor, uint64_t>(state, dst);
    int dstSelectDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstSelectDim);

    TensorInfo<real, uint64_t> srcInfo =
      getTensorInfo<real, THCTensor, uint64_t>(state, src);
    int srcSelectDim = srcInfo.collapseDims(dim);
    srcInfo.reduceDim(srcSelectDim);

    TensorInfo<int64_t, uint64_t> indicesInfo =
      getTensorInfo<int64_t, THCudaLongTensor, uint64_t>(state, indices);
    indicesInfo.collapseDims();

    LARGE_INDEX(real, uint64_t, -1, -1, -1, true);
  }

#undef SMALL_INDEX
#undef LARGE_INDEX
}

#endif
